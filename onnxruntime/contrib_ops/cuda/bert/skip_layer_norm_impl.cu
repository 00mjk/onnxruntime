#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on skipLayerNorm plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/
 
Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/shared_inc/cuda_call.h"
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include <hipcub/hipcub.hpp>
#include "skip_layer_norm_impl.h"

using namespace onnxruntime::cuda;
using namespace hipcub;

namespace onnxruntime {
namespace contrib {
namespace cuda {

/*
 It uses FP16 functions (like hrsqrt and __hadd2), which are only supported on arch >= 5.3
*/
#ifdef USE_CUDA_FP16

template <typename T>
__device__ inline T rsqrt(const T& x);

template <>
__device__ inline float rsqrt(const float& x) {
  return rsqrtf(x);
}

template <>
__device__ inline half rsqrt(const half& x) {
  return hrsqrt(x);
}

struct KeyValuePairSum {
  __device__ inline hipcub::KeyValuePair<float, float> operator()(const hipcub::KeyValuePair<float, float>& a, const hipcub::KeyValuePair<float, float>& b) {
    return hipcub::KeyValuePair<float, float>(a.key + b.key, a.value + b.value);
  }

  __device__ inline hipcub::KeyValuePair<half, half> operator()(const hipcub::KeyValuePair<half, half>& a, const hipcub::KeyValuePair<half, half>& b) {
    const half2 a2 = __halves2half2(a.key, a.value);
    const half2 b2 = __halves2half2(b.key, b.value);
    const half2 res = __hadd2(a2, b2);
    return hipcub::KeyValuePair<half, half>(res.x, res.y);
  }

  __device__ inline hipcub::KeyValuePair<half2, half2> operator()(const hipcub::KeyValuePair<half2, half2>& a, const hipcub::KeyValuePair<half2, half2>& b) {
    return hipcub::KeyValuePair<half2, half2>(__hadd2(a.key, b.key), __hadd2(a.value, b.value));
  }
};

template <typename T, int TPB>
__device__ inline void layerNorm(
    const hipcub::KeyValuePair<T, T>& threadData, const int ld, const int offset, const float* beta, const float* gamma, T* output) {
  // Assuming threadData is already divided by ld

  using BlockReduce = hipcub::BlockReduce<hipcub::KeyValuePair<T, T>, TPB>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ T mu;      // mean
  __shared__ T rsigma;  // 1 / std.dev.

  KeyValuePairSum pairSum;
  const auto sumKV = BlockReduce(temp_storage).Reduce(threadData, pairSum);

  if (threadIdx.x == 0) {
    mu = sumKV.key;
    rsigma = rsqrt(sumKV.value - mu * mu);
  }
  __syncthreads();

  for (int i = threadIdx.x; i < ld; i += TPB) {
    const int idx = offset + i;
    const T val = output[idx];
    const T g(gamma[i]);
    const T b(beta[i]);
    output[idx] = g * (val - mu) * rsigma + b;
  }
}

template <typename T, int TPB>
__device__ inline void layerNormSmall(const T val, const hipcub::KeyValuePair<T, T>& threadData, const int ld, const int idx,
                                      const float* beta, const float* gamma, T* output) {
  // Assuming threadData is already divided by ld
  // Small settings: the block covers the leading dimension TPB >= ld. The input
  // value is available in a register

  using BlockReduce = hipcub::BlockReduce<hipcub::KeyValuePair<T, T>, TPB>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ T mu;      // mean
  __shared__ T rsigma;  // 1 / std.dev.

  KeyValuePairSum pairSum;
  const auto sumKV = BlockReduce(temp_storage).Reduce(threadData, pairSum);

  if (threadIdx.x == 0) {
    mu = sumKV.key;
    rsigma = rsqrt(sumKV.value - mu * mu);
  }
  __syncthreads();

  if (threadIdx.x < ld) {
    const T g(gamma[threadIdx.x]);
    const T b(beta[threadIdx.x]);
    output[idx] = g * (val - mu) * rsigma + b;
  }
}

template <typename T, unsigned TPB>
__global__ void skipLayerNormKernelSmall(
    const int ld, const T* input, const T* skip, const float* beta, const float* gamma, T* output) {
  const T rld = T(1) / T(ld);
  const int offset = blockIdx.x * ld;

  KeyValuePairSum pairSum;
  // reduce x and x^2
  hipcub::KeyValuePair<T, T> threadData(0, 0);
  const int idx = offset + threadIdx.x;
  T val = 0;

  if (threadIdx.x < ld) {
    val = input[idx] + skip[idx];
    const T rldval = rld * val;
    threadData = pairSum(threadData, hipcub::KeyValuePair<T, T>(rldval, rldval * val));
  }

  layerNormSmall<T, TPB>(val, threadData, ld, idx, beta, gamma, output);
}

template <typename T, unsigned TPB>
__global__ void skipLayerNormKernel(
    const int ld, const T* input, const T* skip, const float* beta, const float* gamma, T* output) {
  const T rld = T(1) / T(ld);
  const int offset = blockIdx.x * ld;

  KeyValuePairSum pairSum;
  // reduce x and x^2
  hipcub::KeyValuePair<T, T> threadData(0, 0);

  for (int i = threadIdx.x; i < ld; i += TPB) {
    const int idx = offset + i;
    const T val = input[idx] + skip[idx];
    const T rldval = rld * val;
    threadData = pairSum(threadData, hipcub::KeyValuePair<T, T>(rldval, rldval * val));
    output[idx] = val;
  }

  layerNorm<T, TPB>(threadData, ld, offset, beta, gamma, output);
}

template <typename T>
void computeSkipLayerNorm(hipStream_t stream, const int ld, const int n, const T* input, const T* skip,
                          const float* beta, const float* gamma, T* output) {
  // this must be true because n is the total size of the tensor
  assert(n % ld == 0);
  const int gridSize = n / ld;

  if (ld <= 32) {
    constexpr int blockSize = 32;
    skipLayerNormKernelSmall<T, blockSize>
        <<<gridSize, blockSize, 0, stream>>>(ld, input, skip, beta, gamma, output);
  } else if (ld <= 128) {
    constexpr int blockSize = 128;
    skipLayerNormKernelSmall<T, blockSize>
        <<<gridSize, blockSize, 0, stream>>>(ld, input, skip, beta, gamma, output);
  } else if (ld == 384) {
    constexpr int blockSize = 384;
    skipLayerNormKernelSmall<T, blockSize>
        <<<gridSize, blockSize, 0, stream>>>(ld, input, skip, beta, gamma, output);
  } else {
    constexpr int blockSize = 256;
    skipLayerNormKernel<T, blockSize><<<gridSize, blockSize, 0, stream>>>(ld, input, skip, beta, gamma, output);
  }
  CUDA_CALL(hipPeekAtLastError());
}

void launchSkipLayerNormKernel(
    void* output,
    const void* input,
    const void* skip,
    const float* gamma,
    const float* beta,
    const int batch_size,
    const int hidden_size,
    const int element_count,
    const size_t element_size) {
  // use default stream
  const hipStream_t stream = nullptr;

  if (element_size == 2) {
    computeSkipLayerNorm(stream, hidden_size, element_count,
                         reinterpret_cast<const half*>(input), reinterpret_cast<const half*>(skip),
                         beta, gamma, reinterpret_cast<half*>(output));
  } else {
    computeSkipLayerNorm(stream, hidden_size, element_count,
                         reinterpret_cast<const float*>(input), reinterpret_cast<const float*>(skip),
                         beta, gamma, reinterpret_cast<float*>(output));
  }
}
#endif

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
