#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on embLayerNorm plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/
 
Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "layer_norm.cuh"
#include "embed_layer_norm_impl.h"

using namespace onnxruntime::cuda;
using namespace hipcub;

namespace onnxruntime {
namespace contrib {
namespace cuda {

#ifdef USE_CUDA_FP16

template <unsigned TPB>
__global__ void maskIdxKernelSmall(int sequence_length, const int* mask, int* mask_index) {
  using BlockReduce = hipcub::BlockReduce<int, TPB>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  // blockIdx.x is b
  const int offset = blockIdx.x * sequence_length; // batch strides of sequence_length

  hipcub::Min min;
  int threadData(sequence_length);

  const int idx = offset + threadIdx.x;
  if (threadIdx.x < sequence_length) {
    const int val = mask[idx];
    if (val == 0)  // masked position: report thread idx
    {
      threadData = threadIdx.x;
    }
  }

  const auto min_index = BlockReduce(temp_storage).Reduce(threadData, min);

  if (threadIdx.x == 0) {
    mask_index[blockIdx.x] = min_index;
  }
}

template <unsigned TPB>
__global__ void maskIdxKernel(int sequence_length, const int* mask, int* mask_index) {
  using BlockReduce = hipcub::BlockReduce<int, TPB>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  // blockIdx.x is b
  const int offset = blockIdx.x * sequence_length;  // batch strides of sequence_length

  hipcub::Min min;
  int threadData(sequence_length);

  for (int i = threadIdx.x; i < sequence_length; i += TPB) {
    const int idx = offset + i;
    const int val = mask[idx];
    if (val == 0)  // masked position: report thread idx
    {
      threadData = min(threadData, i);
    }
  }

  const auto min_index = BlockReduce(temp_storage).Reduce(threadData, min);

  if (threadIdx.x == 0) {
    mask_index[blockIdx.x] = min_index;
  }
}

inline int computeMaskIdx(hipStream_t stream, const int sequence_length, const int batch_size, const int* mask, int* mask_index) {
  // Mask idx is of length batch_size and assumes the valid region is contiguous starting
  // from the beginning of the sequence

  // Assume n = batch_size x sequence_length
  if (sequence_length <= 32) {
    maskIdxKernelSmall<32><<<batch_size, 32, 0, stream>>>(sequence_length, mask, mask_index);
  } else if (sequence_length <= 128) {
    maskIdxKernelSmall<128><<<batch_size, 128, 0, stream>>>(sequence_length, mask, mask_index);
  } else if (sequence_length == 384) {
    maskIdxKernelSmall<384><<<batch_size, 384, 0, stream>>>(sequence_length, mask, mask_index);
  } else {
    maskIdxKernel<256><<<batch_size, 256, 0, stream>>>(sequence_length, mask, mask_index);
  }

  CUDA_CALL(hipPeekAtLastError());

  return 0;
}

template <typename T, unsigned TPB>
__global__ void embLayerNormKernel(int hidden_size, const int* input_ids, const int* segment_ids, const float* beta, const float* gamma,
                                   const T* word_embedding, const T* position_embedding, const T* segment_embedding,
                                   T* output) {
  KeyValuePairSum pairSum;
  // 1. lookup word and token of the block
  // blockIdx.x = position in the sequence
  // blockIdx.y = batch
  // gridDim.x = sequence_length
  // gridDim.y = batch_size
  __shared__ int word_id;
  __shared__ int segment_id;

  const T rld = T(1.f) / T(hidden_size);
  const int sequence_position = blockIdx.y * gridDim.x + blockIdx.x;
  if (threadIdx.x == 0) {
    word_id = input_ids[sequence_position];
    segment_id = segment_ids[sequence_position];
  }
  __syncthreads();

  // 2. load pos/tok/word embeddings and add them toghether
  // offset into embeddings is given by word_id * hidden_size
  const int position_offset = blockIdx.x * hidden_size;
  const int word_offset = word_id * hidden_size;
  const int segment_offset = segment_id * hidden_size;
  // the output offset is given by b * (sequence_length * hidden_size) + s * hidden_size
  const int output_offset = sequence_position * hidden_size;

  hipcub::KeyValuePair<T, T> threadData(0, 0);

  for (int it = threadIdx.x; it < hidden_size; it += TPB) {
    const T w(word_embedding[word_offset + it]);
    const T t(segment_embedding[segment_offset + it]);
    const T p(position_embedding[position_offset + it]);
    const T val = w + t + p;

    output[output_offset + it] = val;
    const T rldval = rld * val;
    threadData = pairSum(threadData, hipcub::KeyValuePair<T, T>(rldval, rldval * val));
  }

  // 3. layer norm on the sum
  layerNorm<T, TPB>(threadData, hidden_size, output_offset, beta, gamma, output);
}

template <typename T>
void embSkipLayerNorm(hipStream_t stream, int hidden_size, int batch_size, int sequence_length,
                     const int* input_ids, const int* segment_ids, const float* beta, const float* gamma,
                     const T* word_embedding, const T* position_embedding, const T* segment_embedding,
                     T* output) {
  constexpr int tpb = 256;
  const dim3 grid(sequence_length, batch_size, 1);
  const dim3 block(tpb, 1, 1);

  embLayerNormKernel<T, tpb>
      <<<grid, block, 0, stream>>>(hidden_size, input_ids, segment_ids, beta, gamma, word_embedding, position_embedding, segment_embedding, output);

  CUDA_CALL(hipPeekAtLastError());
}

void launchEmbedLayerNormKernel(void* output,
                                void* mask_index,
                                const int* input_ids,
                                const int* segment_ids,
                                const int* input_mask,
                                const float* gamma,
                                const float* beta,
                                const void* word_embedding,
                                const void* position_embedding,
                                const void* segment_embedding,
                                const int hidden_size,
                                int batch_size,
                                int sequence_length,
                                const size_t element_size) {
  const hipStream_t stream = nullptr; // default stream

  if (element_size == 2) {
    embSkipLayerNorm<half>(stream, hidden_size, batch_size, sequence_length, input_ids, segment_ids,
                           beta, gamma, reinterpret_cast<const half*>(word_embedding), reinterpret_cast<const half*>(position_embedding), reinterpret_cast<const half*>(segment_embedding),
                           reinterpret_cast<half*>(output));
  } else {
    embSkipLayerNorm<float>(stream, hidden_size, batch_size, sequence_length, input_ids, segment_ids,
                           beta, gamma, reinterpret_cast<const float*>(word_embedding), reinterpret_cast<const float*>(position_embedding), reinterpret_cast<const float*>(segment_embedding),
                           reinterpret_cast<float*>(output));
  }

  computeMaskIdx(stream, sequence_length, hidden_size, input_mask, static_cast<int*>(mask_index));
}
#endif
}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
