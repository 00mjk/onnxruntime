#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "topk_impl.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "hipcub/hipcub.hpp"
#include <limits>

namespace onnxruntime {
namespace cuda {

template <typename T>
struct KV {
  T key;
  int64_t val;
};

#define FROM(idx) (left_dim + (idx)*mid_dim + right_dim)
#define TO(idx) (left_dim * K / dimension + (idx)*mid_dim + right_dim)
#define TRIVIAL (1 == largest ? type_min : type_max)
#define BIGGER(n, m) (n.key > m.key ? n : (n.key < m.key ? m : (n.val > m.val ? (1 == largest ? m : n) : (1 == largest ? n : m))))
#define SMALLER(n, m) (n.key < m.key ? n : (n.key > m.key ? m : (n.val < m.val ? (1 == largest ? m : n) : (1 == largest ? n : m))))
#define IS_SMALLER(n, m) (n.key < m.key || !(n.key > m.key) && (1 == largest ? n.val > m.val : n.val < m.val))
#define MAX(n, m) ((n) >= (m) ? (n) : (m))
#define MIN(n, m) ((n) <= (m) ? (n) : (m))

template <typename T>
__global__ void BitonicTopK(const T* X, T* V, int64_t* I, const int64_t* elem_nums, size_t size, int64_t axis, int64_t K, int64_t aligned_K, int64_t largest, int64_t sorted, int64_t dimension, int64_t aligned_dimension, T type_min, T type_max) {
  auto tid = threadIdx.x;
  auto bid = blockIdx.x;
  extern __shared__ char shared_mem[];
  auto S = (KV<T>*)(shared_mem);
  auto mid_dim = axis == size - 1 ? 1 : elem_nums[axis + 1];
  auto left_dim = bid / mid_dim * elem_nums[axis];
  auto right_dim = axis == size - 1 ? 0 : bid % elem_nums[axis + 1];
  //copy x to shared memory
  for (auto i = 0; i < 2; ++i) {
    auto j = (tid << 1) + i;
    if (j < aligned_dimension) {
      S[j].key = j < dimension ? X[FROM(j)] : TRIVIAL;
      S[j].val = j;
    }
  }
  __syncthreads();
  //sort each K
  for (int64_t len = 1; len < aligned_K; len <<= 1) {
    auto dir = len << 1;
    for (auto inc = len; inc > 0; inc >>= 1) {
      auto low = tid & (inc - 1);
      auto i = (tid << 1) - low;
      auto j = i + inc;
      if (j < aligned_dimension) {
        auto reverse = (dir & i) == 0;
        auto swap = reverse ^ IS_SMALLER(S[i], S[j]);
        if (swap) {
          auto tmp = S[i];
          S[i] = S[j];
          S[j] = tmp;
        }
      }
    }
  }
  __syncthreads();
  //merge and rebuild K
  for (int64_t len = aligned_K; len < aligned_dimension; len <<= 1) {
    auto dir = len << 1;
    auto i = (tid << 1) - (tid & (len - 1));
    auto j = i + len;
    if (i % dir < aligned_K && j < aligned_dimension) {
      S[i] = 1 == largest ? BIGGER(S[i], S[j]) : SMALLER(S[i], S[j]);
    }
    __syncthreads();
    for (auto inc = aligned_K >> 1; inc > 0; inc >>= 1) {
      auto ii = (tid << 1) - (tid & (inc - 1));
      auto jj = ii + inc;
      if (ii % dir < aligned_K && jj < aligned_dimension) {
        auto reverse = (dir & ii) == 0;
        auto swap = reverse ^ IS_SMALLER(S[ii], S[jj]);
        if (swap) {
          auto tmp = S[ii];
          S[ii] = S[jj];
          S[jj] = tmp;
        }
      }
    }
    __syncthreads();
  }
  //save top K
  if (1 == sorted) {
    if (1 == largest) {
      auto start = aligned_K - K;
      if (tid >= start && tid < aligned_K) {
        auto to = TO(aligned_K - 1 - tid);
        V[to] = S[tid].key;
        I[to] = S[tid].val;
      }
    } else {
      if (tid < K) {
        auto to = TO(tid);
        V[to] = S[tid].key;
        I[to] = S[tid].val;
      }
    }
  } else {
    if (1 == largest) {
      auto start = aligned_K - K;
      if (tid < start) {
        S[tid].val = aligned_dimension;
      }
    } else {
      if (tid >= K && tid < aligned_K) {
        S[tid].val = aligned_dimension;
      }
    }
    __syncthreads();
    //sort by index ascending
    for (int64_t len = 1; len < aligned_K; len <<= 1) {
      for (int64_t inc = len; inc > 0; inc >>= 1) {
        auto low = tid & (inc - 1);
        auto i = (tid << 1) - low;
        auto j = i + inc;
        if (j < aligned_K && S[i].val > S[j].val) {
          auto tmp = S[i];
          S[i] = S[j];
          S[j] = tmp;
        }
      }
    }
    __syncthreads();
    if (tid < K) {
      auto to = TO(tid);
      V[to] = S[tid].key;
      I[to] = S[tid].val;
    }
  }
}
template <typename T>
__device__ __inline__ bool Equal(const T& t0, const T& t1) {
  auto t2 = t0 > t1 ? t0 - t1 : t1 - t0;
  return (double)t2 < 1.0e-10;
}

template <typename T>
__device__ bool Match(const T* t0, const T* t1, int64_t skip) {
  return (((*t0) ^ (*t1)) >> skip) == 0;
}

__device__ bool Match(const float* f0, const float* f1, int64_t skip) {
  return Match<int32_t>((const int32_t*)f0, (const int32_t*)f1, skip);
}

__device__ bool Match(const double* f0, const double* f1, int64_t skip) {
  return Match<int64_t>((const int64_t*)f0, (const int64_t*)f1, skip);
}

template <typename T>
__device__ bool Test(const T* t, int64_t bit) {
  return ((*t) >> bit) & (T)1;
}

__device__ bool Test(const float* f, int64_t bit) {
  return Test<int32_t>((const int32_t*)f, bit);
}

__device__ bool Test(const double* f, int64_t bit) {
  return Test<int64_t>((const int64_t*)f, bit);
}

template <typename T>
__device__ void Set(T* t, int64_t bit) {
  (*t) |= (T)1 << bit;
}

__device__ void Set(float* f, int64_t bit) {
  Set<int32_t>((int32_t*)f, bit);
}

__device__ void Set(double* f, int64_t bit) {
  Set<int64_t>((int64_t*)f, bit);
}

template <typename T>
__global__ void RadixTopK(const T* X, T* V, int64_t* I, const int64_t* elem_nums, size_t size, int64_t axis, int64_t K, int64_t largest, int64_t sorted, int64_t dimension, int64_t XPT) {
  auto tid = threadIdx.x;
  auto bid = blockIdx.x;
  extern __shared__ char shared_mem[];
  auto S32 = (int32_t*)(shared_mem);
  auto S64 = (int64_t*)(shared_mem);
  auto mid_dim = axis == size - 1 ? 1 : elem_nums[axis + 1];
  auto left_dim = bid / mid_dim * elem_nums[axis];
  auto right_dim = axis == size - 1 ? 0 : bid % elem_nums[axis + 1];
  T Kth = (T)0;
  int32_t global_positive = 0, global_negative = 0;
  auto& thread_positive = S32[tid << 1];
  auto& thread_negative = S32[(tid << 1) + 1];
  thread_positive = thread_negative = 0;
  auto offset = tid * XPT;
  for (int64_t i = 0; i < XPT; ++i) {
    auto j = offset + i;
    if (j < dimension) {
      auto& x = X[FROM(j)];
      if (x > 0) {
        ++thread_positive;
      } else if (x < 0) {
        ++thread_negative;
      }
    }
  }
  __syncthreads();
  for (int64_t i = 2; i < blockDim.x << 1; i <<= 1) {
    auto j = tid * (i << 1);
    auto k = j + i;
    for (int64_t l = 0; l < 2; ++l) {
      auto jj = j + l;
      auto kk = k + l;
      if (kk < blockDim.x << 1) {
        S32[jj] += S32[kk];
      }
    }
  }
  __syncthreads();
  if (0 == tid) {
    global_positive = thread_positive;
    global_negative = thread_negative;
  }
  __syncthreads();
  if (global_positive >= K || K > dimension - global_negative) {
    T sign = 1;
    auto KK = K;
    if (global_positive < KK) {
      sign = -1;
      KK = dimension - KK + 1;
    }
    auto bits = sizeof(T) << 3;
    for (int64_t i = bits - 1; i > -1; --i) {
      S64[tid] = 0;
      for (int64_t j = 0; i < XPT; ++j) {
        auto jj = offset + j;
        if (jj < dimension) {
          T x = sign * X[FROM(jj)];
          if (x > 0 && Match(&x, &Kth, i + 1) && Test(&x, i)) ++S64[tid];
        }
      }
      __syncthreads();
      for (int64_t step = 1; step < blockDim.x; step <<= 1) {
        auto j = tid * (step << 1);
        auto k = j + step;
        if (k < blockDim.x) {
          S64[j] += S64[k];
        }
        __syncthreads();
      }
      if (0 == tid) {
        if (S64[0] >= KK) {
          Set(&Kth, i);
        } else {
          KK -= S64[0];
        }
      }
      __syncthreads();
    }
    __syncthreads();
    if (0 == tid) {
      Kth = (T)sign * Kth;
    }
  }
  if (0 == tid) {
    for (int64_t i = 0, j = 0; i < dimension && j < K; ++i) {
      auto& x = X[FROM(i)];
      if (1 == largest && x > Kth || 0 == largest && x < Kth || Equal(x, Kth)) {
        V[j] = x;
        I[j] = i;
        ++j;
      }
    }
  }
  if (1 == sorted) {
    __syncthreads();
    for (int64_t len = 1; len < K; len <<= 1) {
      for (int64_t inc = len; inc > 0; inc >>= 1) {
        for (int64_t l = 0; l < XPT; ++l) {
          auto t = tid * XPT + l;
          auto low = t & (inc - 1);
          auto i = (t << 1) - low;
          auto j = i + inc;
          if (j < K && (1 == largest && V[i] < V[j] || 0 == largest && V[i] > V[j])) {
            auto vi = V[i];
            V[i] = V[j];
            V[j] = vi;
            auto ii = I[i];
            I[i] = I[j];
            I[j] = ii;
          }
        }
        __syncthreads();
      }
    }
    __syncthreads();
  }
}

template <typename T>
Status TopKImpl(const T* input_x, T* output_v, int64_t* output_i, const int64_t* elem_nums, size_t size, int64_t axis, int64_t K, int64_t largest, int64_t sorted, int64_t N, int64_t dimension) {
  auto aligned_K = static_cast<int64_t>(pow(2, ceil(log2(K))));
  auto aligned_dimension = static_cast<int64_t>(pow(2, ceil(log2(dimension))));
  /*
  if (aligned_dimension <= GridDim::maxThreadsPerBlock << 1) {
    BitonicTopK<T><<<N, GridDim::maxThreadsPerBlock, aligned_dimension * sizeof(KV<T>)>>>(input_x, output_v, output_i, elem_nums, size, axis, K, aligned_K, largest, sorted, dimension, aligned_dimension, std::numeric_limits<T>::min(), std::numeric_limits<T>::max());
    return Status::OK();
  } else {
  */
  auto XPT = static_cast<int64_t>(ceil(static_cast<double>(dimension) / GridDim::maxThreadsPerBlock));
  RadixTopK<T><<<N, GridDim::maxThreadsPerBlock, GridDim::maxThreadsPerBlock * sizeof(int64_t)>>>(input_x, output_v, output_i, elem_nums, size, axis, K, largest, sorted, dimension, XPT);
  return Status::OK();
  //}
}

#define TOPKIMPLE(T) template Status TopKImpl<T>(const T* input_x,         \
                                                 T* output_v,              \
                                                 int64_t* output_i,        \
                                                 const int64_t* elem_nums, \
                                                 size_t size,              \
                                                 int64_t axis,             \
                                                 int64_t K,                \
                                                 int64_t largest,          \
                                                 int64_t sorted,           \
                                                 int64_t N,                \
                                                 int64_t dimension)

TOPKIMPLE(uint8_t);
TOPKIMPLE(uint16_t);
TOPKIMPLE(uint32_t);
TOPKIMPLE(uint64_t);
TOPKIMPLE(int8_t);
TOPKIMPLE(int16_t);
TOPKIMPLE(int32_t);
TOPKIMPLE(int64_t);
TOPKIMPLE(float);
TOPKIMPLE(double);

}  // namespace cuda
}  // namespace onnxruntime
