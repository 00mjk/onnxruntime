#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "orttraining/training_ops/cuda/activation/bias_gelu_grad_impl.h"

#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/shared_inc/fast_divmod.h"
#include "orttraining/training_ops/cuda/activation/gelu_grad_impl_common.cuh"

namespace onnxruntime {
namespace cuda {

namespace {
template <bool use_approximation>
struct GeluGradDxScalarComputer {
  template <typename T>
  __device__ T operator()(const T dY, const T X, const T B) {
    if (use_approximation) {
      return ComputeGeluApproximationGradScalar(dY, X + B);
    } else {
      return ComputeGeluGradScalar(dY, X + B);
    }
  }
};

template <typename T, typename ComputeGeluGradDxScalarFn>
__global__ void BiasGeluGradDxKernel(
    int num_consecutive_elements_per_group, int num_groups_per_thread,
    ComputeGeluGradDxScalarFn compute_gelu_grad_dx_scalar_fn,
    CUDA_LONG input_size, fast_divmod bias_size_fdm,
    const T* dY, const T* X, const T* B, T* dX) {
  const auto& num_threads_per_block = blockDim.x;
  const int& bias_size = bias_size_fdm.d_;
  CUDA_LONG base_idx =
      num_consecutive_elements_per_group * num_groups_per_thread * num_threads_per_block * blockIdx.x +
      num_consecutive_elements_per_group * threadIdx.x;

#pragma unroll
  for (int i = 0; i < num_groups_per_thread; ++i) {
    if (base_idx < input_size) {
      int q, r;
      bias_size_fdm.divmod(base_idx, q, r);
      const int& base_bias_idx = r;

#pragma unroll
      for (int element_idx = 0; element_idx < num_consecutive_elements_per_group; ++element_idx) {
        const int input_idx = base_idx + element_idx;
        if (base_idx < input_size) {
          const int bias_idx =
              base_bias_idx + element_idx - static_cast<int>((base_bias_idx + element_idx) >= bias_size) * bias_size;
          // printf("dX[%d] = GeluGrad(dY[%d], X[%d] + B[%d]); base_bias_idx = %d; bias_size = %d\n", input_idx, input_idx, input_idx, bias_idx, base_bias_idx, bias_size);
          dX[input_idx] = compute_gelu_grad_dx_scalar_fn(dY[input_idx], X[input_idx], B[bias_idx]);
        }
      }

      base_idx += num_consecutive_elements_per_group * num_threads_per_block;
    }
  }
}
}  // namespace

template <typename T>
void LaunchBiasGeluGradDxKernel(
    int64_t input_size, int64_t bias_size,
    const T* dY, const T* X, const T* B, T* dX) {
  constexpr int num_consecutive_elements_per_group = 4;
  constexpr int num_groups_per_thread = 4;
  constexpr int num_threads_per_block = GridDim::maxThreadsPerBlock;
  const auto num_blocks_per_grid = CeilDiv(
      input_size,
      num_threads_per_block * num_consecutive_elements_per_group * num_groups_per_thread);
  const fast_divmod bias_size_fdm{static_cast<int>(bias_size)};
  BiasGeluGradDxKernel<<<num_blocks_per_grid, num_threads_per_block>>>(
      num_consecutive_elements_per_group, num_groups_per_thread,
      GeluGradDxScalarComputer<false>{},
      static_cast<CUDA_LONG>(input_size), bias_size_fdm, dY, X, B, dX);
}

template <typename T>
void LaunchBiasGeluApproximationGradDxKernel(
    int64_t input_size, int64_t bias_size,
    const T* dY, const T* X, const T* B, T* dX) {
  constexpr int num_consecutive_elements_per_group = 4;
  constexpr int num_groups_per_thread = 4;
  constexpr int num_threads_per_block = GridDim::maxThreadsPerBlock;
  const auto num_blocks_per_grid = CeilDiv(
      input_size,
      num_threads_per_block * num_consecutive_elements_per_group * num_groups_per_thread);
  const fast_divmod bias_size_fdm{static_cast<int>(bias_size)};
  BiasGeluGradDxKernel<<<num_blocks_per_grid, GridDim::maxThreadsPerBlock>>>(
      num_consecutive_elements_per_group, num_groups_per_thread,
      GeluGradDxScalarComputer<true>{},
      static_cast<CUDA_LONG>(input_size), bias_size_fdm, dY, X, B, dX);
}

// explicit instantiations
#define SPECIALIZED_BIAS_GELU_GRAD_IMPL(T)   \
  template void LaunchBiasGeluGradDxKernel(  \
      int64_t input_size, int64_t bias_size, \
      const T* dY, const T* X, const T* B, T* dX)

SPECIALIZED_BIAS_GELU_GRAD_IMPL(half);
SPECIALIZED_BIAS_GELU_GRAD_IMPL(float);
SPECIALIZED_BIAS_GELU_GRAD_IMPL(double);

#undef SPECIALIZED_BIAS_GELU_GRAD_IMPL

#define SPECIALIZED_BIAS_GELU_APPROXIMATION_GRAD_IMPL(T) \
  template void LaunchBiasGeluApproximationGradDxKernel( \
      int64_t input_size, int64_t bias_size,             \
      const T* dY, const T* X, const T* B, T* dX)

SPECIALIZED_BIAS_GELU_APPROXIMATION_GRAD_IMPL(half);
SPECIALIZED_BIAS_GELU_APPROXIMATION_GRAD_IMPL(float);
SPECIALIZED_BIAS_GELU_APPROXIMATION_GRAD_IMPL(double);

#undef SPECIALIZED_BIAS_GELU_APPROXIMATION_GRAD_IMPL

}  // namespace cuda
}  // namespace onnxruntime
